#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>
#include<iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
//#include<>
// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

typedef struct {
    int width;
    int heigth;
    int size;
    int *pixel;
}Matrix;

__global__ void Conv(const Matrix input,const Matrix core,const Matrix res) { //_ 每个线程计算一个卷积结果 
    size_t x = threadIdx.x;
    size_t y = threadIdx.y;
    size_t i, j, tmp=0;
    for (j = 0; j < core.heigth; j++) {
        for (i = 0; i < core.width; i++)
            tmp += input.pixel[(y + j) * input.width + x + i] * core.pixel[core.width * j + i]; 
    }
    res.pixel[y * res.width + x] = tmp;
}
//_ 由于所有的线程都使用相同的core,这里可以使用共享内存加载core
//_ 由于卷积结果的相邻点的计算使用了图片的共同像素，应该也可以通过共享内存优化，而且core越大，优化效果越明显，
//_ 是否可以考虑直接把input矩阵加载到共享内存，考虑图片大小和共享内存大小



//_ create host matrix
Matrix createHMatrix(int width, int heigth) {
    Matrix tmp;
    tmp.width = width;
    tmp.heigth = heigth;
    tmp.size = width * heigth * sizeof(int);
    tmp.pixel = (int*)malloc(tmp.size);
    return tmp;
}

//_ create device matrix
Matrix createDMatrix(int width, int heigth) {
    Matrix tmp;
    tmp.width = width;
    tmp.heigth = heigth;
    tmp.size = width * heigth * sizeof(int);
    hipMalloc(&tmp.pixel,tmp.size);
    return tmp;
}

#define IW 5
#define IH 5
#define CW 2
#define CH 2

void initInput(const Matrix &in) {
    for (int i = 0; i < in.heigth; i++) {
        for (int j = 0; j < in.width; j++) {
            in.pixel[i * in.width + j] = i * j;
        }
    }
}

void initCore(const Matrix& core) {
    for (int i = 0; i < core.heigth; i++) {
        for (int j = 0; j < core.width; j++) {
            core.pixel[i * core.width + j] = i + j;
        }
    }
}

void printMatrix(const Matrix &m) {
    std::cout << "w=" << m.width << "\t" << "h=" << m.heigth << std::endl;
    for (int i = 0; i < m.heigth; i++) {
        for (int j = 0; j < m.width; j++) {
            std::cout << m.pixel[i*m.width+j]<<'\t';
        }
        std::cout << std::endl;
    }
}

int main() {
    Matrix inpic, core, outpic;
    Matrix d_inpic, d_core, d_outpic;
    inpic = createHMatrix(IW, IH);
    core = createHMatrix(CW, CH);
    outpic = createHMatrix(IW - CW + 1, IH - CH + 1);
    initInput(inpic);
    initCore(core);

    d_inpic = createDMatrix(IW, IH);
    d_core = createDMatrix(CW, CH);
    d_outpic = createDMatrix(IW - CW + 1, IH - CH + 1);
    hipMemcpy(d_inpic.pixel, inpic.pixel, inpic.size, hipMemcpyHostToDevice);
    hipMemcpy(d_core.pixel, core.pixel, core.size, hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(IW - CW + 1, IH - CH + 1);
    Conv << <grid,block >> > (d_inpic, d_core, d_outpic); //grid为1，block的形状和输出矩阵的形状相同
    hipMemcpy(outpic.pixel, d_outpic.pixel, d_outpic.size, hipMemcpyDeviceToHost);

    printMatrix(outpic);
    hipFree(d_inpic.pixel);
    hipFree(d_core.pixel);
    hipFree(d_outpic.pixel);
}
